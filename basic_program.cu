
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <hipblas.h>

__global__
void fill_in_array(const int n, double *x) {
  const int idx    = threadIdx.x;
  const int stride = blockDim.x;

  for(int i=idx;i<n;i+=stride)
    x[i] = i+1;
}

int main() {

  printf("Beginning C basic program...\n");

  int n = 1024;
  double *x;
  hipMalloc(&x, sizeof(double)*n);

  fill_in_array<<<1, 1>>>(n, x);
  hipError_t err = hipGetLastError();
  if(err) {
    printf("Error name   : %s\n", hipGetErrorName(err));
    printf("Error message: %s\n", hipGetErrorString(err));
  }

  hipblasHandle_t handle;
  hipblasCreate(&handle);

  double sum;
  hipblasDasum(handle, n, x, 1, &sum);

  double exact = n*(n+1)/2.0;

  if( fabs(1.0 - sum/exact) > 1e-15 ) {
    fprintf(stderr, "Sum does not match: %.15e != %.15e\n", sum, exact);
    exit(1);
  }

  hipblasDestroy(handle);
  hipFree(x);

  printf("Finished test with no errors\n");

  return 0;
}
